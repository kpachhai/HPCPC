
#include <hip/hip_runtime.h>
#include <string>
#include <iostream>
#include <fstream>
#include <sstream>
#include <cstdlib>
#include <cstring>
#include <ostream>
#include <sys/time.h>

using namespace std;
    
double get_walltime() {
    struct timeval time;
    if(gettimeofday(&time, NULL)) {
        return 0;
    }
    return (double)time.tv_sec + (double)time.tv_usec * 0.000001;
}

void readPassFile(char **, char ** ,char *);
void readHashFile( char **, char ** ,char *);
void writeFile(string *, string *,  char *, int);

void readPassFile(char * record_pass[], char * record_hash[], char * fileName) {
    int i = 0;
    ifstream file;
    string line;
	
    file.open(fileName); 	//passlist/wordsforsimpletest.txt
    while(getline(file, line)) {
        string key,value;
        istringstream liness(line);
        getline(liness, key, ',');
        getline(liness, value, ',');
        
        char tKey[key.length() + 1];
        char tValue[value.length() + 1];
        strcpy(tKey, key.c_str());
        strcpy(tValue, value.c_str());
        record_pass[i] = (char*) malloc(key.length() + 1);
        record_hash[i] = (char*) malloc(value.length() + 1);
		strncpy(record_pass[i], tKey, key.length()+1);
		strncpy(record_hash[i], tValue, value.length()+1);
        i++;
    }
    file.close();
}

void readHashFile(char ** hToCheck_pass, char ** hToCheck_hash, char * fileName ) {
    ifstream file; 
    string line;
    int i = 0;
    
    file.open(fileName);	//passlist/hashFileToTest.txt
    while(getline(file, line)) {
        string key = "";       
	    string value = line;       
        
        char tKey[key.length() + 1];
        char tValue[value.length() + 1];
        strcpy(tKey, key.c_str());
        strcpy(tValue, value.c_str());
        hToCheck_pass[i] = (char*) malloc(key.length() + 1);
        hToCheck_hash[i] = (char*) malloc(value.length() + 1);
		strncpy(hToCheck_pass[i], tKey, key.length()+1);
		strncpy(hToCheck_hash[i], tValue, value.length()+1);
       i++;       
    }
    file.close();  
}

void writeFile(char ** result_pass, char ** result_hash, char * fileName, int size) { 
	int i;
    ofstream  fileToWriteTo;
    fileToWriteTo.open(fileName); //passlist/convertedHash.txt    
    for(i = 0; i < size; i++) {
        fileToWriteTo << result_pass[i]<< ", " << result_hash[i] << endl;
    }
    fileToWriteTo.close();
}


__device__ int d_strcmp (char * s1, char * s2)
{
	  int ret = 0;
	  while (!(ret = *(unsigned char *) s1 - *(unsigned char *) s2) && *s2) ++s1, ++s2;
	  if (ret < 0)
		ret = -1;
	  else if (ret > 0)
		ret = 1 ;
	  return ret;
}

__global__ void gpuComputation(char ** d_record_pass, char ** d_record_hash, char ** d_hToCheck_pass, char ** d_hToCheck_hash, char ** d_result_pass, char ** d_result_hash, int row) {
	int blockindex = blockIdx.x;
	int threadindex = threadIdx.x;
/*	for(i = 0; i < row; i++) {
		char tempKey[32];
	    for(j = 0; j < 32; j++) {
	    	if(d_record_hash[i][j] == d_hToCheck_hash[threadIdx.x][j]) {
	    		tempKey[j] = d_record_pass[i][j];
	    		printf("h");
	    	}
	    }
*/

   		int res = d_strcmp(d_record_hash[blockindex], d_hToCheck_hash[threadindex]);
   		printf("%c\n", d_record_hash[blockindex][0]);
   		printf("here\n");
	  //  printf("%d %d %d\n", blockindex, threadindex, row);
	    
		if(d_strcmp(d_record_hash[blockindex], d_hToCheck_hash[threadindex]) == 0) {
		    for(int j = 0; j < 32; j++) {
		    	//d_result_pass[threadIdx.x][j] = d_record_pass[blockindex][j];
		    	//d_result_hash[threadIdx.x][j] = d_record_hash[blockindex][j];
		    	printf("%c", d_record_hash[threadIdx.x][j]);
		    }
		    printf("\n");
		}
//	}
}

int performMainComputation(char ** record_pass, char ** record_hash, char ** hToCheck_pass, char ** hToCheck_hash, char ** result_pass, char ** result_hash, int nLinesPFile, int nLinesHFile) {
    int row = nLinesPFile;
    int col = nLinesHFile;

 //   int indexStruct = 0;
 
    int rowSize = row * sizeof(int);
    int colSize = col * sizeof(int);

    char * d_record_pass[nLinesPFile], *d_record_hash[nLinesPFile];
    char * d_hToCheck_pass[nLinesHFile], *d_hToCheck_hash[nLinesHFile];
    char * d_result_pass[nLinesHFile], *d_result_hash[nLinesHFile];
   
   hipMemcpy(d_record_pass, record_pass, rowSize, hipMemcpyHostToDevice);
    hipMemcpy(d_record_hash, record_hash, rowSize, hipMemcpyHostToDevice);
    hipMemcpy(d_hToCheck_pass, hToCheck_pass, rowSize, hipMemcpyHostToDevice);
    hipMemcpy(d_hToCheck_hash, hToCheck_hash, rowSize, hipMemcpyHostToDevice);
    hipMemcpy(d_result_pass, result_pass, rowSize, hipMemcpyHostToDevice);
    hipMemcpy(d_result_hash, result_hash, rowSize, hipMemcpyHostToDevice);
        
    dim3 blocks(row, 1);
    dim3 grids(col, 1);
    
    for(int i = 0; i < 16; i++)
    	cout << record_pass[i] << " " << d_record_pass[i] << endl;
    
    gpuComputation<<<grids, blocks, 1>>>(d_record_pass, d_record_hash, d_hToCheck_pass, d_hToCheck_hash, d_result_pass, d_result_hash, row);
    hipDeviceSynchronize();
    
    hipMemcpy(result_pass, d_result_pass, colSize, hipMemcpyDeviceToHost);
    hipMemcpy(result_hash, d_result_hash, colSize, hipMemcpyDeviceToHost); 

   hipFree(d_record_pass);	hipFree(d_hToCheck_pass);	hipFree(d_result_pass);
   hipFree(d_record_hash);	 hipFree(d_hToCheck_hash);	hipFree(d_result_hash);   
  
/*
    for(i = 0; i < row; i++) {
        for(j = 0; j < col; j++) {
            if(record[i].hash.compare(hToCheck[j].hash) == 0) {
				result[indexStruct].pass = record[i].pass;
				result[indexStruct].hash = record[i].hash;
				indexStruct++;
            }
		}
    }
*/
  return 16;
}


void printBenchmark(int nLinesHFile, double readTime, int nPassCracked, double execTime, double writeTime) {
    cout << endl;
    cout << "Read time of the file with " << nLinesHFile << " pass hashes = " << readTime << " seconds" << endl << endl;
    cout << "Total number of passwords cracked = " << nPassCracked << endl;
    cout << "Total execution time for the main computation = " << execTime << " seconds" << endl;
    cout << endl;
    cout << "Write time of the output file = " << writeTime << " seconds" << endl << endl;
}

int main(int argc, char ** argv) {
    char * PFile = argv[1];
    int nLinesPFile = atoi(argv[2]);
    char * HFile = argv[3];
    int nLinesHFile = atoi(argv[4]);
    char * outputFile = argv[5]; 

    char * record_pass[nLinesPFile];
    char * record_hash[nLinesPFile];
    char * hToCheck_pass[nLinesHFile];
    char * hToCheck_hash[nLinesHFile];
    char * result_pass[nLinesHFile];
    char * result_hash[nLinesHFile];
        
    double startReadTime = get_walltime();   
    readPassFile(record_pass, record_hash, PFile);
    readHashFile(hToCheck_pass, hToCheck_hash, HFile);
    double readTime = get_walltime() - startReadTime;
 
        
    double startExecTime = get_walltime();
    int nPassCracked = performMainComputation(record_pass, record_hash, hToCheck_pass, hToCheck_hash, result_pass, result_hash, nLinesPFile, nLinesHFile);
    double execTime = get_walltime() - startExecTime;


    double startWriteTime = get_walltime();
//    writeFile(result_pass, result_hash, outputFile, nPassCracked);
    double writeTime = get_walltime() - startWriteTime;

    //printBenchmark(nLinesHFile, readTime, nPassCracked, execTime, writeTime);
  
    return 0;
    
}

