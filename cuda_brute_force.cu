#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>

#include <iostream>
#include <fstream>
#include <sstream>
#include <cstring>
#include <ostream>

// #include <time.h>
#include "md5.h"

#define HASH_LENGTH 33

int forceCrack(char* hash, int maxPassLength); // Cracks the password
void checkPass(char* hash, char* string1); // Compares two hashes
// void forceCrackIn( int * spot_, char * str_, int maxPassLength_, bool *search_, char hash_);

__global__ void forceCrackIn( int * spot_, char * str_, int maxPassLength_, bool *search_, char * alphanum_, int alphanumLength_)
{
	int j, k;
	
	printf("Hello thread %d\n", threadIdx.x);
	if(threadIdx.x == 0){
		printf("%d\n", maxPassLength_);
		printf("%d\n", alphanumLength_);
		printf("%s\n", alphanum_[0]);
	}

	for(j = 0; j < maxPassLength_; j++) 				// For each character in the string
	{
		printf("a from thread %d\n", threadIdx.x);
		printf("spot_[j] = %d from thread %d\n",spot_[j], threadIdx.x);
		if(spot_[j] == alphanumLength_)				// If we're at the last character
		{
			printf("b from thread %d\n", threadIdx.x);
			spot_[j] = 0;						// Reset back to the first
			for(k = 1; k < maxPassLength_; k++)		// And "carry" to the right (Multiple times if multiple carries)
			{
				printf("c from thread %d\n", threadIdx.x);
				if (j+k == maxPassLength_)
				{
					printf("d from thread %d\n", threadIdx.x);
					search_ = false;
					break;
				}
				if(spot_[j+k] < alphanumLength_)
				{
					printf("e from thread %d\n", threadIdx.x);
					spot_[j+k] += 1;
					break;						// Successful carry. Stop.
				}
			}
		}
		if(spot_[j] >= 0)
		{ 
			printf("f from thread %d\n", threadIdx.x);
			printf("alphanum_[spot_[j]] is %c\n", alphanum_[spot_[j]]);
			str_[j] = alphanum_[spot_[j]];
			printf("str_[j] is %c\n", str_[j]);

		}

	} 							// Increment least significant character
}


MD5 md5; // Yes, these are a global variable

int main(int argc, char **argv)
{

	int maxPassLength = 5; // Will not check passwords longer than this
	char hash[33];


	sprintf(hash, "%s",argv[1]);

	std::cout << "Pre force crack" << std::endl;
    forceCrack(hash, maxPassLength); // This does the brute force
    std::cout << "Done!" << std::endl;

    return 0;
}

int forceCrack(char* hash, int maxPassLength)
{
	std::cout << "In force crack" << std::endl;
	char alphanum[62];  							// Character set to check passwords of
	std::cout << "1" << std::endl;

	sprintf(alphanum, "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789");
	std::cout << "2" << std::endl;
    // std::cout << alphanum[0] << std::endl;

	int alphanumLength = strlen(alphanum); 		// 62
	// std::cout << alphanumLength << std::endl;
	std::cout << "3" << std::endl;

	char * temp = alphanum;

	int spot[maxPassLength]; 					// Gives you the index into alphanum you should have for each character place
	char str[maxPassLength];					// Test string
	std::cout << "5" << std::endl;

	bool search = true;

	std::cout << "Init Spot and String" << std::endl;
	for (int i = 0; i < maxPassLength; i++) {		// Set initial values for arrays
		spot[i] = -1;
		str[i] = 0;
	}
	spot[0] = 0;

    //---------------------//
	// Cuda Version
	//---------------------//
	// varibles that will need to be accessed by kernel are copied to gpu
	int * spotPtr;
	char  * strPtr;
	const int spotSize = maxPassLength*sizeof(int);
	const int strSize = maxPassLength*sizeof(char);


	// Allocate and Transfer memory to the device
	std::cout << "Starting Cuda Stuff" << std::endl;
	hipMalloc( (void**)&spotPtr, spotSize ); 
	hipMalloc( (void**)&strPtr, strSize );
	
	hipMemcpy( spotPtr, spot, spotSize, hipMemcpyHostToDevice ); 
	hipMemcpy( strPtr, str, strSize, hipMemcpyHostToDevice );

	// Perform the array addition
	// dimBlock defines number of threads kernel will run on
	dim3 dimBlock( maxPassLength );
	// dimGrid defines number of blocks scheduled on different compute units on gpu  
	dim3 dimGrid ( 1 );
	// while (search)
	// {
	std::cout << "Cracking" << std::endl;
		// forceCrackIn( int * spot_, char * str_, int maxPassLength_, bool *search_, char * alphanum_, int alphanumLength_)
	forceCrackIn<<<dimGrid, dimBlock>>>(spotPtr, strPtr, maxPassLength, &search, temp, alphanumLength);
	std::cout << "Done Cracking" << std::endl;
		// Copy the Contents from the GPU
		checkPass(hash, str);				// Checks the string
		spot[0]++; 	
	// }
		hipMemcpy( str, strPtr, strSize, hipMemcpyDeviceToHost );
		hipFree( str );

    // CUDA VERSION DONE //


		return 0;
	}


	void checkPass(char* hash, char* tempString)
	{
		char testHash[HASH_LENGTH];
		int i;
		bool foundPass = false;
		sprintf(testHash, "%s", md5.digestString(tempString));

		printf("%s\n", tempString);
	for (i = 0; i < HASH_LENGTH; i++) {// Checks if the hashes match
		if (testHash[i] != hash[i]) {
			foundPass = false;
			break;
		} else {
			foundPass = true;
		}
	}
	if (foundPass) {
		printf("The password is %s\n", tempString);
		exit(0);
	}
}
